#include "hip/hip_runtime.h"
#include "visualTur_device.hpp"
#include "FreeImage.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <sys/time.h>
#include "stdlib.h"

int main(int argc, char ** argv)
{
	if (argc < 4)
	{
		std::cerr<<"Error, testVisualTur hdf5_file dataset_name octree_file [device]"<<std::endl;
		return 0;
	}

	int device = 0;
	if (argc > 4)
	{
		device = atoi(argv[4]);
		hipSetDevice(device);
	}

	int W = 1024;
	int H = 1024;

	visualTurParams_device_t params;
	params.W = W;
	params.H = H;
	params.fov_H = 35.0f;
	params.fov_W = 35.0f;
	params.distance = 50.0f;
	params.numRayPx = 1;
	params.maxElementsCache = 35;
	params.maxElementsCache_CPU = 500;
	params.dimCubeCache = make_int3(128,128,128);
	params.cubeInc = 3;
	params.levelCubes = 2;
	params.octreeLevel =5;
	params.hdf5File = argv[1];
	params.dataset_name = argv[2];
	params.octreeFile = argv[3];

	params.numThreads = 1;
	params.deviceID = device;
	params.startRay = 0;
	params.endRay = params.W*params.H*params.numRayPx;

	float * screenG = 0;
	float * screenC = new float[H*W*4];

	for(int i=0; i<H; i++)
		for(int j=0; j<W; j++)
		{
			int id = i*W + j;
			screenC[id*4] = 0.0;
			screenC[id*4+1] = 0.0f;
			screenC[id*4+2]= 0.0f;
			screenC[id*4+3]= 0.0f;
		} 

	std::cerr<<"Allocating memory octree CUDA screen: "<< hipGetErrorString(hipMalloc((void**)&screenG, sizeof(float)*H*W*4))<<std::endl;

	std::cerr<<"Cuda mem set: "<<hipGetErrorString(hipMemset((void *)screenG,0,sizeof(float)*H*W*4))<<std::endl;		

	visualTur_device * VisualTur= new visualTur_device(params, screenG); 
//	VisualTur->camera_Move(make_float3(1500.0f, 128.0f, 4100.0f));
	VisualTur->camera_Move(make_float3(128.0f, 128.0f, 550.0f));
	//VisualTur->camera_Move(make_float3(1500.0f, 128.0f, 550.0f));
	//VisualTur->camera_MoveForward(1.0f);

	VisualTur->updateVisibleCubes();

	FreeImage_Initialise();
	FIBITMAP * bitmap = FreeImage_Allocate(H,W,24);
	RGBQUAD color;
#if 0
	struct timeval st, end;
	gettimeofday(&st, NULL);
	for(int m=0; m<10; m++)
	{ 
		for(int i=0; i<H; i++)
			for(int j=0; j<W; j++)
			{
				int id = i*W + j;
				screenC[id*4] = 0.0;
				screenC[id*4+1] = 0.0f;
				screenC[id*4+2]= 0.0f;
				screenC[id*4+3]= 0.0f;
			} 

		VisualTur->updateVisibleCubes();

		std::cerr<<"Retrieve screen from GPU: "<< hipGetErrorString(hipMemcpy((void*) screenC, (const void*) screenG, sizeof(float)*W*H*4, hipMemcpyDeviceToHost))<<std::endl;

		int hits =0;
		for(int i=0; i<H; i++)
			for(int j=0; j<W; j++)
			{
				int id = i*W + j;
				if (screenC[id*4]!=0.0f || screenC[id*4+1]!=0.0f || screenC[id*4+2]!=0.0f)
					hits++;
				color.rgbRed = screenC[id*4]*255;
				color.rgbGreen = screenC[id*4+1]*255;
				color.rgbBlue = screenC[id*4+2]*255;
				FreeImage_SetPixelColor(bitmap, i, j, &color);
			} 
		std::cout<<"--->"<<hits<<std::endl;
		std::stringstream name;
		name<<"prueba"<<m<<".png";
		FreeImage_Save(FIF_PNG, bitmap, name.str().c_str(), 0);
		VisualTur->camera_StrafeRight(0.5f);
	}
	gettimeofday(&end, NULL);
	double delta = ((end.tv_sec  - st.tv_sec) * 1000000u + end.tv_usec - st.tv_usec) / 1.e6;
	std::cout << "Time elapsed: " << delta << " sec"<< std::endl;

	hipFree(screenG);
	delete[] screenC;
	FreeImage_DeInitialise();
#endif
	delete VisualTur;
}
