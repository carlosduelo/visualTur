#include "hip/hip_runtime.h"
#include "visualTur_device.hpp"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>

visualTur_device::visualTur_device(visualTurParams_device_t initParams, float * p_pixelBuffer)
{
	// Create octree
	octreeLevel 	= initParams.octreeLevel;
	octree 		= new Octree_device(initParams.octreeFile, octreeLevel);

	numThreads 	= initParams.numThreads;
	deviceThreads 	= new visualTur_thread*[numThreads];
	deviceID	= initParams.deviceID;

	// Create Cache
	cache 		= new lruCache_device(initParams.hdf5File, initParams.dataset_name, initParams.maxElementsCache, initParams.dimCubeCache, initParams.cubeInc, initParams.levelCubes, initParams.octreeLevel, initParams.maxElementsCache_CPU);

	pixelBuffer 		= p_pixelBuffer;

	

	visualTurParams_thread_t initParams_thread;
	initParams_thread.W 			= initParams.W;
	initParams_thread.H 			= initParams.H;
	initParams_thread.distance		= initParams.distance;
	initParams_thread.fov_H 		= initParams.fov_H;
	initParams_thread.fov_W 		= initParams.fov_W;
	initParams_thread.numRayPx 		= initParams.numRayPx;

	// Cube Cache settings
	initParams_thread.levelCubes 		= initParams.levelCubes;

	// hdf5 settings
	initParams_thread.hdf5File 		= initParams.hdf5File;
	initParams_thread.dataset_name 		= initParams.dataset_name;

	// Octree
	initParams_thread.octreeLevel 		= initParams.octreeLevel;

	initParams_thread.device		= deviceID;


	int totalRays 	= initParams.endRay - initParams.startRay;
	int numRays 	= totalRays / numThreads;
	int modRays	= totalRays % numThreads;

	initParams_thread.startRay      = initParams.startRay;
        initParams_thread.endRay        = initParams.startRay + numRays + modRays;
        deviceThreads[0] 		= new visualTur_thread(initParams_thread, octree, cache, pixelBuffer);

	for(int i=1; i<numThreads; i++)
	{
		initParams_thread.startRay 	= initParams_thread.endRay;
		initParams_thread.endRay 	= initParams_thread.startRay + numRays;
		deviceThreads[i] = new visualTur_thread(initParams_thread, octree, cache, pixelBuffer + (4*initParams_thread.startRay));
	}

}

visualTur_device::~visualTur_device()
{
	for(int i=0; i<numThreads; i++)
		delete deviceThreads[i];

	delete octree;
	delete[] deviceThreads;
	delete cache;
}
		
void	visualTur_device::camera_Move(float3 Direction)
{
	
	for(int i=0; i<numThreads; i++)
		deviceThreads[i]->camera_Move(Direction);

	void * status;
	for(int i=0; i<numThreads; i++)
	{
		int rc = pthread_join(deviceThreads[i]->getID_thread(), &status);
                if (rc)
                {
                        std::cerr << "Error:unable to join," << rc << std::endl;
                        exit(-1);
                }
	}
}

void	visualTur_device::camera_RotateX(float Angle)
{
	
	for(int i=0; i<numThreads; i++)
		deviceThreads[i]->camera_RotateX(Angle);

	void * status;
	for(int i=0; i<numThreads; i++)
	{
		int rc = pthread_join(deviceThreads[i]->getID_thread(), &status);
                if (rc)
                {
                        std::cerr << "Error:unable to join," << rc << std::endl;
                        exit(-1);
                }
	}
}

void	visualTur_device::camera_RotateY(float Angle)
{
	
	for(int i=0; i<numThreads; i++)
		deviceThreads[i]->camera_RotateY(Angle);

	void * status;
	for(int i=0; i<numThreads; i++)
	{
		int rc = pthread_join(deviceThreads[i]->getID_thread(), &status);
                if (rc)
                {
                        std::cerr << "Error:unable to join," << rc << std::endl;
                        exit(-1);
                }
	}
}

void	visualTur_device::camera_RotateZ(float Angle)
{
	
	for(int i=0; i<numThreads; i++)
		deviceThreads[i]->camera_RotateZ(Angle);

	void * status;
	for(int i=0; i<numThreads; i++)
	{
		int rc = pthread_join(deviceThreads[i]->getID_thread(), &status);
                if (rc)
                {
                        std::cerr << "Error:unable to join," << rc << std::endl;
                        exit(-1);
                }
	}
}

void	visualTur_device::camera_MoveForward(float Distance)
{
	
	for(int i=0; i<numThreads; i++)
		deviceThreads[i]->camera_MoveForward(Distance);

	void * status;
	for(int i=0; i<numThreads; i++)
	{
		int rc = pthread_join(deviceThreads[i]->getID_thread(), &status);
                if (rc)
                {
                        std::cerr << "Error:unable to join," << rc << std::endl;
                        exit(-1);
                }
	}
}

void	visualTur_device::camera_MoveUpward(float Distance)
{
	
	for(int i=0; i<numThreads; i++)
		deviceThreads[i]->camera_MoveUpward(Distance);

	void * status;
	for(int i=0; i<numThreads; i++)
	{
		int rc = pthread_join(deviceThreads[i]->getID_thread(), &status);
                if (rc)
                {
                        std::cerr << "Error:unable to join," << rc << std::endl;
                        exit(-1);
                }
	}
}

void	visualTur_device::camera_StrafeRight(float Distance)
{
	
	for(int i=0; i<numThreads; i++)
		deviceThreads[i]->camera_StrafeRight(Distance);

	void * status;
	for(int i=0; i<numThreads; i++)
	{
		int rc = pthread_join(deviceThreads[i]->getID_thread(), &status);
                if (rc)
                {
                        std::cerr << "Error:unable to join," << rc << std::endl;
                        exit(-1);
                }
	}
}


void visualTur_device::updateVisibleCubes()
{
	
	for(int i=0; i<numThreads; i++)
		deviceThreads[i]->updateVisibleCubes();
	
	void * status;
	for(int i=0; i<numThreads; i++)
	{
		int rc = pthread_join(deviceThreads[i]->getID_thread(), &status);
                if (rc)
                {
                        std::cerr << "Error:unable to join," << rc << std::endl;
                        exit(-1);
                }
	}
}

