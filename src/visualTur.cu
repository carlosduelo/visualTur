#include "hip/hip_runtime.h"
#include "visualTur.hpp"
#include <iostream>
#include <fstream>

visualTur::visualTur(visualTurParams_t initParams)
{
	// Creating Camera
	camera = new Camera(initParams.numRayPx, initParams.H, initParams.W, initParams.distance, initParams.fov_H, initParams.fov_W);

	// Creating visible cubes array
	visibleCubesCPU = new visibleCube_t[camera->get_numRays()];
	std::cerr<<"Allocating memory visibleCubesGPU: "<< hipGetErrorString(hipMalloc((void**)&visibleCubesGPU, camera->get_numRays()*sizeof(visibleCube_t)))<<std::endl;
	resetVisibleCubes();

	// Cache creation
	cache = new lruCache(initParams.hdf5File, initParams.dataset_name, initParams.maxElementsCache, initParams.dimCubeCache, initParams.cubeInc);

	// Create octree
	octree = new Octree(initParams.octreeFile, camera);

	// Create rayCaster
	raycaster = new rayCaster(octree->getIsosurface(), make_float3(0.0f, 512.0f, 0.0f));

}

visualTur::~visualTur()
{
	delete 		camera;
	delete		cache;
	delete[]	visibleCubesCPU;
	hipFree(visibleCubesGPU);
	delete		octree;
	delete		raycaster;
}

void visualTur::resetVisibleCubes()
{
	int max = camera->get_numRays();
	for(int i=0; i<max; i++)
	{
		visibleCubesCPU[i].id = 0;
		visibleCubesCPU[i].data = 0;
	}
	std::cerr<<"Coping visibleCubes CPU to GPU: "<< hipGetErrorString(hipMemcpy((void*)visibleCubesGPU, (const void*)visibleCubesCPU, camera->get_numRays()*sizeof(visibleCube_t), hipMemcpyHostToDevice))<<std::endl;

}

void visualTur::changeScreen(int pW, int pH, float pfovW, float pfovH, float pDistance)
{
	camera->set_W(pW);
	camera->set_H(pH);
	camera->set_fovW(pfovW);
	camera->set_fovH(pfovH);
	camera->set_Distance(pDistance);

	delete[]	visibleCubesCPU;
	hipFree(visibleCubesGPU);
	visibleCubesCPU = new visibleCube_t[camera->get_numRays()];
	std::cerr<<"Allocating memory visibleCubesGPU: "<< hipGetErrorString(hipMalloc((void**)&visibleCubesGPU, camera->get_numRays()*sizeof(visibleCube_t)))<<std::endl;
	resetVisibleCubes();
}

void visualTur::changeNumRays(int pnR)
{
	camera->set_RayPerPixel(pnR);

	delete[]	visibleCubesCPU;
	hipFree(visibleCubesGPU);
	visibleCubesCPU = new visibleCube_t[camera->get_numRays()];
	std::cerr<<"Allocating memory visibleCubesGPU: "<< hipGetErrorString(hipMalloc((void**)&visibleCubesGPU, camera->get_numRays()*sizeof(visibleCube_t)))<<std::endl;
	resetVisibleCubes();
}

void visualTur::changeCacheParameters(int nE, int3 cDim, int cInc)
{
	cache->changeDimensionCube(nE, cDim, cInc);

	delete[]	visibleCubesCPU;
	hipFree(visibleCubesGPU);
	visibleCubesCPU = new visibleCube_t[camera->get_numRays()];
	std::cerr<<"Allocating memory visibleCubesGPU: "<< hipGetErrorString(hipMalloc((void**)&visibleCubesGPU, camera->get_numRays()*sizeof(visibleCube_t)))<<std::endl;
	resetVisibleCubes();
}

void	visualTur::camera_Move(float3 Direction)
{
	camera->Move(Direction);
}
void	visualTur::camera_RotateX(float Angle)
{
	camera->RotateX(Angle);
}
void	visualTur::camera_RotateY(float Angle)
{
	camera->RotateY(Angle);
}
void	visualTur::camera_RotateZ(float Angle)
{
	camera->RotateZ(Angle);
}
void	visualTur::camera_MoveForward(float Distance)
{
	camera->MoveForward(Distance);
}
void	visualTur::camera_MoveUpward(float Distance)
{
	camera->MoveUpward(Distance);
}
void	visualTur::camera_StrafeRight(float Distance)
{
	camera->StrafeRight(Distance);
}

void visualTur::updateVisibleCubes(int level, float * pixelBuffer)
{
	octree->getBoxIntersected(level, visibleCubesGPU, visibleCubesCPU);

	#if 1
	int hits = 0;
	for(int i=0; i<camera->get_numRays(); i++)
		if (visibleCubesCPU[i].id != 0)
			hits++;
	std::cout<<"Hits "<<hits<<std::endl;
	#endif

	cache->updateCache(visibleCubesCPU, camera->get_numRays(), octree->getnLevels());

	std::cerr<<"Coping visibleCubes to GPU: "<<hipGetErrorString(hipMemcpy((void*) visibleCubesGPU, (const void*) visibleCubesCPU, camera->get_numRays()*sizeof(visibleCube_t), hipMemcpyHostToDevice))<<std::endl;

	raycaster->render(camera, level, octree->getnLevels(), visibleCubesGPU, cache->get_cubeDim(), make_int3(cache->get_cubeInc(),cache->get_cubeInc(),cache->get_cubeInc()), pixelBuffer); 
}
