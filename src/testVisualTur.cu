#include "hip/hip_runtime.h"
#include "visualTur.hpp"
#include "FreeImage.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include "stdlib.h"

int main(int argc, char ** argv)
{
	if (argc < 4)
	{
		std::cerr<<"Error, testVisualTur hdf5_file dataset_name octree_file [device]"<<std::endl;
		return 0;
	}

	int device = 0;
	if (argc > 4)
	{
		device = atoi(argv[4]);
		hipSetDevice(device);
	}

	visualTurParams_t params;
	params.W = 800;
	params.H = 800;
	params.fov_H = 35.0f;
	params.fov_W = 35.0f;
	params.distance = 50.0f;
	params.numRayPx = 1;
	params.maxElementsCache = 1000;
	params.dimCubeCache = make_int3(32,32,32);
	params.cubeInc = 2;
	params.hdf5File = argv[1];
	params.dataset_name = argv[2];
	params.octreeFile = argv[3];

	visualTur * VisualTur = new visualTur(params); 

	VisualTur->camera_Move(make_float3(52.0f, 52.0f, 520.0f));
	VisualTur->camera_MoveForward(1.0f);

	FreeImage_Initialise();

	float * screenG = 0;
	float * screenC = new float[800*800*4];

	for(int i=0; i<800; i++)
		for(int j=0; j<800; j++)
		{
			int id = i*800 + j;
			screenC[id*4] = 0.0;
			screenC[id*4+1] = 0.0f;
			screenC[id*4+2]= 0.0f;
			screenC[id*4+3]= 0.0f;
		} 
	
	FIBITMAP * bitmap = FreeImage_Allocate(800,800,24);
	RGBQUAD color;

	std::cerr<<"Allocating memory octree CUDA screen: "<< hipGetErrorString(hipMalloc((void**)&screenG, sizeof(float)*800*800*4))<<std::endl;

	std::cerr<<"Cuda mem set: "<<hipGetErrorString(hipMemset((void *)screenG,0,sizeof(float)*800*800*4))<<std::endl;		


	for(int it=0; it<10; it++)
	{
		VisualTur->updateVisibleCubes(5, screenG);

		std::cerr<<"Retrieve screen from GPU: "<< hipGetErrorString(hipMemcpy((void*) screenC, (const void*) screenG, sizeof(float)*800*800*4, hipMemcpyDeviceToHost))<<std::endl;

		int hits =0;
		for(int i=0; i<800; i++)
			for(int j=0; j<800; j++)
			{
				int id = i*800 + j;
				if (screenC[id*4]!=0.0f || screenC[id*4+1]!=0.0f || screenC[id*4+2]!=0.0f)
					hits++;
				/*
				picture[i*3] = screenC[i*4]*255;
				picture[i*3+1] = screenC[i*4+1]*255;
				picture[i*3+2] = screenC[i*4+2]*255;
				color.rgbRed = 0;
				color.rgbGreen = (double)i/800*255.0;
				color.rgbBlue = (double)j/800*255.0;
				*/
				color.rgbRed = screenC[id*4]*255;
				color.rgbGreen = screenC[id*4+1]*255;
				color.rgbBlue = screenC[id*4+2]*255;
				FreeImage_SetPixelColor(bitmap, i, j, &color);
			} 
		std::cout<<"--->"<<hits<<std::endl;
		std::stringstream name;
		name<<"prueba"<<it<<".png";
		FreeImage_Save(FIF_PNG, bitmap, name.str().c_str(), 0);
	}

	hipFree(screenG);
	delete[] screenC;
	FreeImage_DeInitialise();
}
